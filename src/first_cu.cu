#include "hip/hip_runtime.h"
#include "first_cu.h"

#include <thrust/sort.h>

//kernal definition
__global__ void mat_add(){

}

FirstCuda::FirstCuda() {}

FirstCuda::~FirstCuda() {}

void FirstCuda::run_cuda_op() {
	check_cuda_dev(0);
}

void FirstCuda::check_cuda_dev(int dev_id){
	hipDeviceProp_t dev_prop;
	hipGetDeviceProperties(&dev_prop, dev_id);
	std::cout << "使用GPU device " << dev_id << ": " << dev_prop.name << std::endl;
	std::cout << "SM的数量：" << dev_prop.multiProcessorCount << std::endl;
	std::cout << "每个线程块的共享内存大小：" << dev_prop.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
	std::cout << "每个线程块的最大线程数：" << dev_prop.maxThreadsPerBlock << std::endl;
	std::cout << "每个EM的最大线程数：" << dev_prop.maxThreadsPerMultiProcessor << std::endl;
	std::cout << "每个SM的最大线程束数：" << dev_prop.maxThreadsPerMultiProcessor / 32 << std::endl;
}